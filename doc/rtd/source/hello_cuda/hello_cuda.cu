#include <hamr_buffer.h>
#include <hamr_buffer_util.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <memory>

#include "add.h"
#include "write.h"

int main(int, char **)
{
    size_t n_vals = 400;

    // allocate and initialize to 1 on the GPU
    hamr::buffer<float> a0(hamr::buffer_allocator::cuda, n_vals, 1.0f);

    // allocate and initialize to 1 on the host
    hamr::buffer<float> a1(hamr::buffer_allocator::malloc, n_vals, 1.0f);

    // add the two arrays
    hamr::buffer<float> a2 = add(a0, a1);

    // write the result
    write(std::cerr, a2);

    return 0;
}
