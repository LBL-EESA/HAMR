#include "hamr_cuda_uva_memory_resource.h"

#include <iostream>

namespace hamr
{

// --------------------------------------------------------------------------
p_cuda_uva_memory_resource cuda_uva_memory_resource::New()
{
    return std::shared_ptr<cuda_uva_memory_resource>(new cuda_uva_memory_resource);
}

// --------------------------------------------------------------------------
p_memory_resource cuda_uva_memory_resource::new_instance() const
{
    return std::shared_ptr<cuda_uva_memory_resource>(new cuda_uva_memory_resource);
}

// --------------------------------------------------------------------------
void *cuda_uva_memory_resource::do_allocate(std::size_t n_bytes, std::size_t align)
{
    (void) align;

    void *ptr = nullptr;

    hipError_t ierr = hipMallocManaged(&ptr, n_bytes, hipMemAttachGlobal);
    if (ierr != hipSuccess)
    {
        std::cerr << "[" << __FILE__ << ":" << __LINE__ << "] Error:"
            << "Failed to allocate " << n_bytes << " of CUDA managed memory. "
            << hipGetErrorString(ierr) << std::endl;

        throw std::bad_alloc();
    }

    if (this->verbose > 1)
    {
        std::cerr << "cuda_uva_memory_resource(" << this << ") allocated " << n_bytes
            << " alligned to " << align << " byte boundary at " << ptr << std::endl;
    }

    return ptr;
}

// --------------------------------------------------------------------------
void cuda_uva_memory_resource::do_deallocate(void *ptr, std::size_t n_bytes,
    std::size_t align)
{
    (void) n_bytes;
    (void) align;

    hipError_t ierr = hipFree(ptr);

    if (ierr != hipSuccess)
    {
        std::cerr << "[" << __FILE__ << ":" << __LINE__ << "] Error:"
            << "Failed to free " << n_bytes << " of CUDA managed memory at "
            << ptr << ". " << hipGetErrorString(ierr) << std::endl;
    }

    if (this->verbose > 1)
    {
        std::cerr << "cuda_uva_memory_resource(" << this << ") deallocated " << n_bytes
            << " alligned to " << align << " byte boundary  at " << ptr << std::endl;
    }
}

// --------------------------------------------------------------------------
bool cuda_uva_memory_resource::do_is_equal(const pmr_memory_resource& other) const noexcept
{
    return dynamic_cast<const cuda_uva_memory_resource*>(&other) != nullptr;
}

}
